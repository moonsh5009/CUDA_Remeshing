#include "ProjectiveDynamics.cuh"

void ProjectiveDynamics::initProject(Cloth* cloths) {
	initProject_kernel << <divup(cloths->_numNodes, MAX_BLOCKSIZE), MAX_BLOCKSIZE >> > (
		*cloths->_param, *cloths->_pdParam);
	CUDA_CHECK(hipPeekAtLastError());
}
void ProjectiveDynamics::compErrorProject(Cloth* cloths) {
#if CLOTH_STRAIN==0
	compEdgeErrorProject_kernel << <divup(cloths->_edgeConstraints->_numConstraints, BLOCKSIZE), BLOCKSIZE >> > (
		*cloths->_param, *cloths->_pdParam, *cloths->_edgeConstraints);
	CUDA_CHECK(hipPeekAtLastError());
#else
	compTriangleErrorProject_kernel << <divup(cloths->_triangleConstraints->_numConstraints, BLOCKSIZE), BLOCKSIZE >> > (
		*cloths->_param, *cloths->_pdParam, *cloths->_triangleConstraints);
	CUDA_CHECK(hipPeekAtLastError());
#endif
#if CLOTH_BENDING==0
	compEdgeErrorProject_kernel << <divup(cloths->_edgeBendingConstraints->_numConstraints, BLOCKSIZE), BLOCKSIZE >> > (
		*cloths->_param, *cloths->_pdParam, *cloths->_edgeBendingConstraints);
	CUDA_CHECK(hipPeekAtLastError());
#else
	compDihedralErrorProject_kernel << <divup(cloths->_dihedralBendingConstraints->_numConstraints, BLOCKSIZE), BLOCKSIZE >> > (
		*cloths->_param, *cloths->_pdParam, *cloths->_dihedralBendingConstraints);
	CUDA_CHECK(hipPeekAtLastError());
#endif

}
void ProjectiveDynamics::updateXsProject(
	Cloth* cloths, REAL* maxError)
{
	REAL* d_maxError;
	CUDA_CHECK(hipMalloc((void**)&d_maxError, sizeof(REAL)));
	CUDA_CHECK(hipMemset(d_maxError, 0, sizeof(REAL)));

	updateXsProject_kernel << <divup(cloths->_numNodes, BLOCKSIZE), BLOCKSIZE, BLOCKSIZE * sizeof(REAL) >> > (
		*cloths->_param, *cloths->_pdParam, d_maxError);
	CUDA_CHECK(hipPeekAtLastError());

	CUDA_CHECK(hipMemcpy(maxError, d_maxError, sizeof(REAL), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipFree(d_maxError));
}